
#include<stdio.h>
#include<hip/hip_runtime.h>
#define row1 2 /* Number of rows of first matrix */
#define col1 3 /* Number of columns of first matrix */
#define row2 3 /* Number of rows of second matrix */
#define col2 2 /* Number of columns of second matrix */

__global__ void matproductsharedmemory(int *l,int *m, int *n)
{
    int x=blockIdx.x;
    int y=blockIdx.y;
    __shared__ int p[col1];

    int i;
    int k=threadIdx.x;

    n[col2*y+x]=0;

   p[k]=l[col1*y+k]*m[col2*k+x];

  __syncthreads();

  for(i=0;i<col1;i++)
  n[col2*y+x]=n[col2*y+x]+p[i];
}

int main()
{
    int a[row1][col1];
    int b[row2][col2];
    int c[row1][col2];
    int *d,*e,*f;
    int i,j;

    printf("\nElements of first matrix of size 2*3\n");
    for(i=0;i<row1;i++)
    {
        for(j=0;j<col1;j++)
            {
                a[i][j] = i+j+1;
                printf("%d ",a[i][j]);
            }
            printf("\n");
    }
    printf("\nElements of second matrix of size 3*2\n");
        for(i=0;i<row2;i++)
        {
            for(j=0;j<col2;j++)
                { 

                b[i][j] = i+j+2;
                    printf("%d ",b[i][j]);
                }
                printf("\n");
        }

   hipMalloc((void **)&d,row1*col1*sizeof(int));
    hipMalloc((void **)&e,row2*col2*sizeof(int));
    hipMalloc((void **)&f,row1*col2*sizeof(int));

 hipMemcpy(d,a,row1*col1*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(e,b,row2*col2*sizeof(int),hipMemcpyHostToDevice);

dim3 grid(col2,row1);

matproductsharedmemory<<<grid,col1>>>(d,e,f);

 hipMemcpy(c,f,row1*col2*sizeof(int),hipMemcpyDeviceToHost);

 printf("\n Product of two matrices:\n");
    for(i=0;i<row1;i++)
    {
        for(j=0;j<col2;j++)
        {
              printf("%d\t",c[i][j]);
        }
        printf("\n");
    }

    hipFree(d);
    hipFree(e);
    hipFree(f);

    return 0;
}