
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void arradd(int *x,int *y,int *z){
	int index=blockIdx.x;
  z[index]=x[index]+y[index];
  printf("\nElements at index %d : %d + %d = %d",index,x[index],y[index],z[index]);
}

int main(){

	int n=3;
	int a[3]={5,8,9};
	int b[3]={10,11,12};
	int c[n];
	int *x,*y,*z;
	int i;
	
	hipMalloc((void**)&x,n*sizeof(int));
	hipMalloc((void**)&y,n*sizeof(int));
	hipMalloc((void**)&z,n*sizeof(int));
	hipMemcpy(x,a,n*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(y,b,n*sizeof(int),hipMemcpyHostToDevice);
  arradd<<<n,1>>>(x,y,z);
   
    hipMemcpy(c,z,n*sizeof(int),hipMemcpyDeviceToHost);
    printf("\nResultant vector:");
    for(i=0;i<n;i++){
    	printf("%d ",c[i]);
    }
    printf("\n");
    hipFree(x);
    hipFree(y);
    hipFree(z);
    return 0;

}